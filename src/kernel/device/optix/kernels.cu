#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/helpers.h>

#include "types.h"

extern "C" {
    __constant__ ParamsForTest params;
}

extern "C"
__global__ void __raygen__fixed() {
    uint3 launch_idx = optixGetLaunchIndex();
    Pixel* data = reinterpret_cast<Pixel*>(optixGetSbtDataPointer());
    params.pixels[launch_idx.y * params.image_width + launch_idx.x] =
        make_color(make_float3(data->r, data->g, data->b));
}